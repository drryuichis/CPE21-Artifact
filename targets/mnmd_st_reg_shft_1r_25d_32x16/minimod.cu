#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <float.h>
#include <math.h>
#include <time.h>

#include "../../constants.h"
#include "../../grid.h"

#define N_RADIUS 4
#define N_THREADS_X_DIM 32
#define N_THREADS_Y_DIM 16
#define N_THREADS_Z_DIM 0

// Constant memory coefficients
__constant__ float c_coef0;
__constant__ float c_coefx[N_RADIUS+1];
__constant__ float c_coefy[N_RADIUS+1];
__constant__ float c_coefz[N_RADIUS+1];

__global__ void __launch_bounds__(1024) kernel_1r_25d(
    llint nx, llint ny, llint nz, int ldimx, int ldimy, int ldimz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    float *__restrict__ phi, const float *__restrict__ eta
) {
    __shared__ float s_u[N_THREADS_Y_DIM+2*N_RADIUS][N_THREADS_X_DIM+2*N_RADIUS];

    const llint j0 = blockIdx.y * blockDim.y;
    const llint k0 = blockIdx.x * blockDim.x;

    const llint je = min(j0 + N_THREADS_Y_DIM, ny);
    const llint ke = min(k0 + N_THREADS_X_DIM, nz);

    const llint j = j0 + threadIdx.y;
    const llint k = k0 + threadIdx.x;

    const llint sje = (j0+N_THREADS_Y_DIM<ny) ? N_THREADS_Y_DIM : ((ny-1)%N_THREADS_Y_DIM+1);
    const llint ske = (k0+N_THREADS_X_DIM<nz) ? N_THREADS_X_DIM : ((nz-1)%N_THREADS_X_DIM+1);

    const llint suj = threadIdx.y + N_RADIUS;
    const llint suk = threadIdx.x + N_RADIUS;

    float infront1, infront2, infront3, infront4; // variables for input "in front of" the current slice
    float behind1, behind2, behind3, behind4; // variables for input "behind" the current slice
    float current; // input value in the current slice

    behind3  = u[IDX3(-4,j,k)];
    behind2  = u[IDX3(-3,j,k)];
    behind1  = u[IDX3(-2,j,k)];
    current  = u[IDX3(-1,j,k)];
    infront1 = u[IDX3(0,j,k)];
    infront2 = u[IDX3(1,j,k)];
    infront3 = u[IDX3(2,j,k)];
    infront4 = u[IDX3(3,j,k)];

    for (llint i = 0; i < nx; i++) {
        // advance the slice (move the thread-front)
        behind4  = behind3;
        behind3  = behind2;
        behind2  = behind1;
        behind1  = current;
        current  = infront1;
        infront1 = infront2;
        infront2 = infront3;
        infront3 = infront4;
        infront4 = u[IDX3(i+N_RADIUS,j,k)];

        __syncthreads();

        if (threadIdx.y < N_RADIUS) {
            s_u[threadIdx.y][suk] = u[IDX3(i, j - N_RADIUS, k)];
            s_u[threadIdx.y+sje+N_RADIUS][suk] = u[IDX3(i, threadIdx.y+je, k)];
        }
        if (threadIdx.x < N_RADIUS) {
            s_u[suj][threadIdx.x] = u[IDX3(i,j,k - N_RADIUS)];
            s_u[suj][threadIdx.x+ske+N_RADIUS] = u[IDX3(i,j,threadIdx.x+ke)];
        }

        s_u[suj][suk] = u[IDX3(i,j,k)];

        __syncthreads();

        if (j < ny && k < nz) {
            float lap = __fmaf_rn(c_coef0, current
                      , __fmaf_rn(c_coefx[1], __fadd_rn(infront1,behind1)
                      , __fmaf_rn(c_coefy[1], __fadd_rn(s_u[suj+1][suk],s_u[suj-1][suk])
                      , __fmaf_rn(c_coefz[1], __fadd_rn(s_u[suj][suk+1],s_u[suj][suk-1])
                      , __fmaf_rn(c_coefx[2], __fadd_rn(infront2,behind2)
                      , __fmaf_rn(c_coefy[2], __fadd_rn(s_u[suj+2][suk],s_u[suj-2][suk])
                      , __fmaf_rn(c_coefz[2], __fadd_rn(s_u[suj][suk+2],s_u[suj][suk-2])
                      , __fmaf_rn(c_coefx[3], __fadd_rn(infront3,behind3)
                      , __fmaf_rn(c_coefy[3], __fadd_rn(s_u[suj+3][suk],s_u[suj-3][suk])
                      , __fmaf_rn(c_coefz[3], __fadd_rn(s_u[suj][suk+3],s_u[suj][suk-3])
                      , __fmaf_rn(c_coefx[4], __fadd_rn(infront4,behind4)
                      , __fmaf_rn(c_coefy[4], __fadd_rn(s_u[suj+4][suk],s_u[suj-4][suk])
                      , __fmul_rn(c_coefz[4], __fadd_rn(s_u[suj][suk+4],s_u[suj][suk-4])
            )))))))))))));

            if (i >= x3 && i < x4 && j >= y3 && j < y4 && k >= z3 && k < z4) {
                v[IDX3(i,j,k)] = __fmaf_rn(2.f, current,
                    __fmaf_rn(vp[IDX3(i,j,k)], lap, -v[IDX3(i,j,k)])
                );
            } else {
                const float s_eta_c = eta[IDX3(i,j,k)];

                v[IDX3(i,j,k)] = __fdiv_rn(
                    __fmaf_rn(
                        __fmaf_rn(2.f, s_eta_c,
                            __fsub_rn(2.f,
                                __fmul_rn(s_eta_c, s_eta_c)
                            )
                        ),
                        current,
                        __fmaf_rn(
                            vp[IDX3(i,j,k)],
                            __fadd_rn(lap, phi[IDX3(i,j,k)]),
                            -v[IDX3(i,j,k)]
                        )
                    ),
                    __fmaf_rn(2.f, s_eta_c, 1.f)
                );

                phi[IDX3(i,j,k)] = __fdiv_rn(
                        __fsub_rn(
                            phi[IDX3(i,j,k)],
                            __fmaf_rn(
                            __fmul_rn(
                                __fsub_rn(eta[IDX3(i+1,j,k)], eta[IDX3(i-1,j,k)]),
                                __fsub_rn(infront1,behind1)
                            ), hdx_2,
                            __fmaf_rn(
                            __fmul_rn(
                                __fsub_rn(eta[IDX3(i,j+1,k)], eta[IDX3(i,j-1,k)]),
                                __fsub_rn(s_u[suj+1][suk], s_u[suj-1][suk])
                            ), hdy_2,
                            __fmul_rn(
                                __fmul_rn(
                                    __fsub_rn(eta[IDX3(i,j,k+1)], eta[IDX3(i,j,k-1)]),
                                    __fsub_rn(s_u[suj][suk+1], s_u[suj][suk-1])
                                ),
                            hdz_2)
                            ))
                        )
                    ,
                    __fadd_rn(1.f, s_eta_c)
                );
            }
        }
    }
}

__global__ void kernel_add_source_kernel(float *g_u, llint idx, float source) {
    g_u[idx] += source;
}

extern "C" void target(
    uint nsteps, double *time_kernel,
    const grid_t grid,
    llint sx, llint sy, llint sz,
    float hdx_2, float hdy_2, float hdz_2,
    const float *__restrict__ coefx, const float *__restrict__ coefy, const float *__restrict__ coefz,
    float *__restrict__ u, const float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta, const float *__restrict__ source
) {
    struct timespec start, end;

    float *d_u = allocateDeviceGrid(grid);
    float *d_v = allocateDeviceGrid(grid);
    float *d_phi = allocateDeviceGrid(grid);
    float *d_eta = allocateDeviceGrid(grid);
    float *d_vp = allocateDeviceGrid(grid);

    hipMemset (d_u, 0, gridSize(grid));
    hipMemset (d_v, 0, gridSize(grid));
    hipMemcpy(d_vp, vp, gridSize(grid), hipMemcpyDefault);
    hipMemcpy(d_phi, phi, gridSize(grid), hipMemcpyDefault);
    hipMemcpy(d_eta, eta, gridSize(grid), hipMemcpyDefault);

    float coef0 = coefx[0] + coefy[0] + coefz[0];
    hipMemcpyToSymbol(HIP_SYMBOL(c_coef0), &coef0, sizeof (float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_coefx), coefx, (N_RADIUS + 1) * sizeof (float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_coefy), coefy, (N_RADIUS + 1) * sizeof (float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_coefz), coefz, (N_RADIUS + 1) * sizeof (float));

    dim3 threadsPerBlock(N_THREADS_X_DIM, N_THREADS_Y_DIM, 1);

    const uint npo = 100;
    for (uint istep = 1; istep <= nsteps; ++istep) {
        clock_gettime(CLOCK_REALTIME, &start);

        dim3 n_block_front(
            (grid.nz+N_THREADS_X_DIM-1) / N_THREADS_X_DIM,
            (grid.ny+N_THREADS_Y_DIM-1) / N_THREADS_Y_DIM);
        kernel_1r_25d<<<n_block_front, threadsPerBlock, 0>>>(
            grid.nx,grid.ny,grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            grid.x3,grid.x4,grid.y3,grid.y4,grid.z3,grid.z4,
            grid.lx,grid.ly,grid.lz,
            hdx_2, hdy_2, hdz_2,
            d_u, d_v, d_vp,
            d_phi, d_eta);

        kernel_add_source_kernel<<<1, 1>>>(d_v, IDX3_grid(sx,sy,sz,grid), source[istep-1]);
        clock_gettime(CLOCK_REALTIME, &end);
        *time_kernel += (end.tv_sec  - start.tv_sec) +
                        (double)(end.tv_nsec - start.tv_nsec) / 1.0e9;

        float *t = d_u;
        d_u = d_v;
        d_v = t;

        // Print out
        if (istep % npo == 0) {
            printf("time step %u / %u\n", istep, nsteps);
        }
    }

    hipMemcpy(u, d_u, gridSize(grid), hipMemcpyDeviceToHost);

    freeDeviceGrid(d_u, grid);
    freeDeviceGrid(d_v, grid);
    freeDeviceGrid(d_vp, grid);
    freeDeviceGrid(d_phi, grid);
    freeDeviceGrid(d_eta, grid);
}
